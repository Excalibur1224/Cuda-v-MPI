#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdint.h>
#include <unistd.h>
#include <sys/types.h>
#include <unistd.h>
#include <fcntl.h>
#include <sys/time.h>
#include <errno.h>
#include <time.h>
#include <stdio.h>
#include <cmath>

#include "ex4vel.h"

#define SP 32
#define SM 2 //run with less than 3 SM's for linear interpolation
#define STEPS_PER_SEC 10000
#define STEPS 1000000000 //1 billion steps


__device__ double table_accel(int timeidx, double *d_DefaultProfile)
{
    long unsigned int tsize = sizeof(d_DefaultProfile) / sizeof(double);

    // Check array bounds for look-up table
    if(timeidx > tsize)
    {
        // printf("timeidx=%d exceeds table size = %lu and range %d to %lu\n", timeidx, tsize, 0, tsize-1);
    }

    return d_DefaultProfile[timeidx];
}

__device__ double faccel(double time, double *d_DefaultProfile)
{
    int timeidx = (int)time;
    int timeidx_next = ((int)time)+1;
    double delta_t = time - (double)((int)time);

    return (table_accel(timeidx, d_DefaultProfile) + ( (table_accel(timeidx_next, d_DefaultProfile) - 
            table_accel(timeidx, d_DefaultProfile)) * delta_t));
}


__global__ void cuda_function(double *d_sums, double start, double end, int sp, int sm){
    int my_rank = threadIdx.x; // gets rank for block
	int my_block = blockIdx.x; // gets block id to calculate rank
	double localstart, localend, a, b, n;
	my_rank += sp*my_block; // calculate rank across all blocks
    int workers = sp*sm; //calculate total workers

    localstart = (my_rank*(end/workers));
    localend = ((my_rank*(end/workers))+(end/workers)); //calculate subranges

    a = localstart;
    b = localend;
    n = STEPS/workers;

    double h = (b - a) / n;
    double sum = 0.0;

    // printf("thread %d with %lf to %lf\n", my_rank, localstart, localend);

    for(int i=0;i<n;++i){ //riemann sum for given range
        double x = a + (i*h);
        double fx = sin(x);
        sum += fx;
    }
    d_sums[my_rank] = (h*sum); //assign rank's sum to proper array index to be copied to host program
}

__global__ void cuda_test(double *d_InterpProfile, double *d_DefaultProfile, double *d_sums, double a, double b, int sp, int sm){
	int my_rank = threadIdx.x;
	int my_block = blockIdx.x;
	int localstart, localend;
	int n = sp*sm;
	my_rank += sp*my_block;

    localstart =((my_rank*(1800/n))*STEPS_PER_SEC);
    localend=(((my_rank*(1800/n))+(1800/n))*STEPS_PER_SEC);

    // printf("thread %d active\n", my_rank);

    double time, dt, sum;
    dt = 1.0/STEPS_PER_SEC;
    for(int i=localstart;i<localend;i++){
        // time you would use in your integrator and faccel(time) is the fuction to integrate
        time = 0.0 + (dt*(double)i);
        d_InterpProfile[i] = faccel(time, d_DefaultProfile);
    }

    for(int i=localstart;i<localend;i++){
        sum += d_InterpProfile[i]; //integrate off of interpolated profile
    }
    d_sums[my_rank] = sum/STEPS_PER_SEC;
}


int main(){
    struct timespec start, stop;
    double fstart, fstop;
    clock_gettime(CLOCK_MONOTONIC, &start); fstart=(double)start.tv_sec + ((double)start.tv_nsec/1000000000.0);
	double *h_InterpProfile; //declares arrays to be copied between host and device
	double *d_InterpProfile;
    double *d_DefaultProfile;
    double *h_sums;
    double *d_sums;
	int N = SP*SM;
    h_sums = (double*)malloc(sizeof(double)*N); //allocate space on host
    d_sums = (double*)malloc(sizeof(double)*N);
	h_InterpProfile = (double*)malloc(sizeof(double)*STEPS_PER_SEC*1800); //will not throw error if incorrect size,
	d_InterpProfile = (double*)malloc(sizeof(double)*STEPS_PER_SEC*1800); //and will simply return less than expected.

	hipMalloc((void**)&d_InterpProfile, STEPS_PER_SEC*sizeof(double)*1800); //allocate space on device
    hipMalloc((void**)&d_DefaultProfile, sizeof(double)*1800);
    hipMalloc((void**)&d_sums, sizeof(double)*N);

	hipMemcpy(d_InterpProfile,h_InterpProfile,STEPS_PER_SEC*sizeof(double)*1800,hipMemcpyHostToDevice); //initializes data
    hipMemcpy(d_DefaultProfile,DefaultProfile,sizeof(double)*1800,hipMemcpyHostToDevice);
    hipMemcpy(d_sums,h_sums,sizeof(double)*N,hipMemcpyHostToDevice);

	dim3 grid_size(SM); //delcares cuda threads to be passed in
	dim3 block_size(SP);

	cuda_test<<<grid_size,block_size>>>(d_InterpProfile, d_DefaultProfile, d_sums, 0, 1800, SP, SM); //calls kernel
    // cuda_function<<<grid_size,block_size>>>(d_sums, 0, M_PI, SP, SM);

	hipDeviceSynchronize(); //waits for all threads to complete

	hipMemcpy(h_InterpProfile,d_InterpProfile,STEPS_PER_SEC*sizeof(double)*1800,hipMemcpyDeviceToHost);
    hipMemcpy(h_sums,d_sums,sizeof(double)*N,hipMemcpyDeviceToHost); //copy new device data back to host

    double gsum;
    for(int i=0;i<N;i++){
        gsum += h_sums[i]; //total all sums for results
    }
    clock_gettime(CLOCK_MONOTONIC, &stop); fstop=(double)stop.tv_sec + ((double)stop.tv_nsec/1000000000.0);
    printf("%lf seconds\n", (fstop-fstart));
    printf("final distance is:%lf\n", gsum);

	hipFree(d_InterpProfile); //frees all device allocated memory
    hipFree(d_DefaultProfile);
    hipFree(d_sums);
	free(h_InterpProfile); //frees all host memory
    free(h_sums);

	return 0;
}